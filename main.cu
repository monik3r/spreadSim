
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DSIZE 1024*1024
#define nTPB 256

/***********************/
/* CUDA ERROR CHECKING */
/***********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess){
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line); 
        if (abort) exit(code);
    }
}

/*************************/
/* CURAND INITIALIZATION */
/*************************/
__global__ void initCurand(hiprandState *state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

/**********************/
/*  CURAND GENERATION */
/**********************/
__global__ void genUniform(unsigned long seed, float *a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    a[idx] = hiprand_uniform(&state);
}

__global__ void genNormal(unsigned long seed, float *a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    a[idx] = hiprand_normal(&state);
}

/********/
/* MAIN */
/********/
int main() {
    hiprandState *devState;  gpuErrchk(hipMalloc((void**)&devState, DSIZE*sizeof(hiprandState)));
    float *d_a;             gpuErrchk(hipMalloc((void**)&d_a, DSIZE*sizeof(float)));

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    initCurand<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, 1);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    genUniform<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(1, d_a);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Initialization time:  %3.1f ms \n", time);

}